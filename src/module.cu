#include "hip/hip_runtime.h"
#include "../include/module.h"
#include "../include/timer.h"
#include "../include/cuda_check.h"
#include "../include/cuda_rand.h"
#include "../include/gpu_params.h"
#include <cmath>
#include <iostream>

// ################################################################################################################
/**
 * Dense matrix multiplication layer. 
*/
Matmul::Matmul(CudaVariable *cuda_a, CudaVariable *cuda_b, CudaVariable *cuda_c, int m, int n, int p) : 
        cuda_a(cuda_a), cuda_b(cuda_b), cuda_c(cuda_c), m(m), n(n), p(p) {}

__global__ void matmul_forward_parallel(float *a, float *b, float *c, int m, int n, int p, int TILE_SIZE) {
    // Multiplication of matrices A and B; the result is stored in the matrix C
    extern __shared__ float a_tile[], b_tile[];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < m && j < p) {
        float sum = 0.0f;
        for (int k = 0; threadIdx.x + k < n; k += TILE_SIZE) {
            a_tile[threadIdx.y * n + threadIdx.x + k] = a[i * n + threadIdx.x + k];
            b_tile[(threadIdx.y + k) * p + threadIdx.x] = b[(threadIdx.y + k) * p + j];
        }
        __syncthreads();
        for (int k = 0; k < n; ++k)
            sum += a_tile[i * n + k] * b_tile[k * p + j];
        c[i * p + j] = sum;
    }
}

void Matmul::forward(bool training) {
    timer_start(TMR_MATMUL_FW);
    cuda_c->zero();
    // GPU blocks and threads settings
    dim3 blocksPerGrid((m + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, (p + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_2D, MAX_THREADS_PER_BLOCK_2D, 1); // 2D tiles
    // Launch kernel
    int sharedMemorySize = 2 * MAX_THREADS_PER_BLOCK_2D * n;
    if (sharedMemorySize * sizeof(float) > SHARED_MEMORY_PER_BLOCK)
        std::cerr << "The size of the data exceeds the size of available shared memory per block." << std::endl;
    matmul_forward_parallel<<<blocksPerGrid, threadsPerBlock, sharedMemorySize * sizeof(float)>>>(cuda_a->data, cuda_b->data, cuda_c->data, m, n, p, MAX_THREADS_PER_BLOCK_2D);
    check_kernel_call();
    hipDeviceSynchronize();
   /*
    c->zero();
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < p; k++)
                c->data[i * p + k] += a->data[i * n + j] * b->data[j * p + k];
        }
    */
    timer_stop(TMR_MATMUL_FW);
}

__global__ void matmul_backward_parallel(float *a_data, float *b_data, float *a_grad, float *b_grad, float *c_grad, int m, int n, int p) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < m && j < n) {
        float tmp = 0;
        for (int k = 0; k < p; k++) {
            tmp += c_grad[i * p + k] * b_data[j * p + k];
            atomicAdd(&b_grad[j * p + k], c_grad[i * p + k] * a_data[i * n + j]); // TRY TO AVOID ATOMIC ADD
        }
		a_grad[i * n + j] = tmp;
    }
}

void Matmul::backward() {
    timer_start(TMR_MATMUL_BW);
    cuda_a->zero_grad();
    cuda_b->zero_grad();
    // GPU blocks and threads settings
    dim3 blocksPerGrid((m + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, (n + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_2D, MAX_THREADS_PER_BLOCK_2D, 1);
    // Launch kernel
    matmul_backward_parallel<<<blocksPerGrid, threadsPerBlock>>>(cuda_a->data, cuda_b->data, cuda_a->grad, cuda_b->grad, cuda_c->grad, m, n, p);
    check_kernel_call();
    hipDeviceSynchronize();
   /*
    a->zero_grad();
    b->zero_grad();
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++) {
            float tmp = 0;
            for (int k = 0; k < p; k++) {
                tmp += c->grad[i * p + k] * b->data[j * p + k];
                b->grad[j * p + k] += c->grad[i * p + k] * a->data[i * n + j];
            }
		    a->grad[i * n + j] = tmp;
        }
    */
    timer_stop(TMR_MATMUL_BW);
}


// ################################################################################################################

/**
 * A sparse matrix multiplication layer.
*/
SparseMatmul::SparseMatmul(CudaVariable *cuda_a, CudaVariable *cuda_b, CudaVariable *cuda_c, SparseIndex *sp, int m, int n, int p) :
        cuda_a(cuda_a), cuda_b(cuda_b), cuda_c(cuda_c), m(m), n(n), p(p) {
            CudaSparseIndex *cuda_sp_temp = new CudaSparseIndex(sp->indices.data(), sp->indptr.data(), sp->indices.size(), sp->indptr.size());
            cuda_sp = cuda_sp_temp;
        }

__global__ void sparsematmul_forward_parallel(float *a, float *b, float *c, int *indptr, int *indices, int N, int p) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int k = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < N && k < p) {
        for (int jj = indptr[i]; jj < indptr[i + 1]; jj++) {
            int j = indices[jj];
            c[i * p + k] += a[jj] * b[j * p + k];
        }
    }        
}

void SparseMatmul::forward(bool training) {
    timer_start(TMR_SPMATMUL_FW);
    cuda_c->zero();
    // GPU blocks and threads settings
    dim3 blocksPerGrid((m + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, (p + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_2D, MAX_THREADS_PER_BLOCK_2D, 1);
    // Launch kernel
    sparsematmul_forward_parallel<<<blocksPerGrid, threadsPerBlock>>>(cuda_a->data, cuda_b->data, cuda_c->data, cuda_sp->indptr, cuda_sp->indices, cuda_sp->indptr_size - 1, p);
    check_kernel_call();
    hipDeviceSynchronize();
    /*
    c->zero();
    for (int i = 0; i < sp->indptr.size() - 1; i++)
        for (int jj = sp->indptr[i]; jj < sp->indptr[i + 1]; jj++) {
            int j = sp->indices[jj];
            for (int k = 0; k < p; k++)
                c->data[i * p + k] += a->data[jj] * b->data[j * p + k];
        }
    */
    timer_stop(TMR_SPMATMUL_FW);
}

__global__ void sparsematmul_backward_parallel(float *a_data, float *b_grad, float *c_grad, int *indptr, int *indices, int N, int p) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int k = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < N && k < p) {
        for (int jj = indptr[i]; jj < indptr[i + 1]; jj++) {
            int j = indices[jj];
            atomicAdd(&b_grad[j * p + k], c_grad[i * p + k] * a_data[jj]); // TRY TO AVOID ATOMIC ADD
        }
    }
}

void SparseMatmul::backward() {
    timer_start(TMR_SPMATMUL_BW);
    cuda_b->zero_grad();
    // GPU blocks and threads settings
    dim3 blocksPerGrid((m + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, (p + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_2D, MAX_THREADS_PER_BLOCK_2D, 1);
    // Launch kernel
    sparsematmul_backward_parallel<<<blocksPerGrid, threadsPerBlock>>>(cuda_a->data, cuda_b->grad, cuda_c->grad, cuda_sp->indptr, cuda_sp->indices, cuda_sp->indptr_size - 1, p);
    check_kernel_call();
    hipDeviceSynchronize();
    /*
    b->zero_grad();
    for (int i = 0; i < sp->indptr.size() - 1; i++)
        for (int jj = sp->indptr[i]; jj < sp->indptr[i + 1]; jj++) {
            int j = sp->indices[jj];
            for (int k = 0; k < p; k++)
                    b->grad[j * p + k] += c->grad[i * p + k] * a->data[jj];
        }
    */
    timer_stop(TMR_SPMATMUL_BW);
}

// ################################################################################################################

/**
 * A specialized sparse matrix multiplication for graphs.
*/
GraphSum::GraphSum(CudaVariable *cuda_in, CudaVariable *cuda_out, SparseIndex *graph, int dim) :
        cuda_in(cuda_in), cuda_out(cuda_out), dim(dim) {
            CudaSparseIndex *cuda_graph_temp = new CudaSparseIndex(graph->indices.data(), graph->indptr.data(), graph->indices.size(), graph->indptr.size());
            cuda_graph = cuda_graph_temp;
        }

__global__ void graphsum_forward_parallel(float *in, float *out, int *indptr, int *indices, int N, int dim) {
    int src = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (src < N && j < dim) {
        for (int i = indptr[src]; i < indptr[src + 1]; i++) {
            int dst = indices[i];
            float coef = 1.0 / sqrtf(
                    (indptr[src + 1] - indptr[src]) * (indptr[dst + 1] - indptr[dst])
            );
            // This only works for undirected graphs. Should be out[dst] += coef * in[src]
            out[src * dim + j] += coef * in[dst * dim + j];
        }
    }        
}

void GraphSum::forward(bool training) {
    timer_start(TMR_GRAPHSUM_FW);
    cuda_out->zero();
    // GPU blocks and threads settings
    dim3 blocksPerGrid((cuda_graph->indptr_size - 1 + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, (dim + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_2D, MAX_THREADS_PER_BLOCK_2D, 1);
    // Launch kernel
    graphsum_forward_parallel<<<blocksPerGrid, threadsPerBlock>>>(cuda_in->data, cuda_out->data, cuda_graph->indptr, cuda_graph->indices, cuda_graph->indptr_size - 1, dim);
    check_kernel_call();
    hipDeviceSynchronize();
    /*
    out->zero();
    for (int src = 0; src < graph->indptr.size() - 1; src++)
        for (int i = graph->indptr[src]; i < graph->indptr[src + 1]; i++) {
            int dst = graph->indices[i];
            float coef = 1.0 / sqrtf(
                    (graph->indptr[src + 1] - graph->indptr[src]) * (graph->indptr[dst + 1] - graph->indptr[dst])
            );
            for (int j = 0; j < dim; j++)
                // This only works for undirected graphs. Should be out[dst] += coef * in[src]
                out->data[src * dim + j] += coef * in->data[dst * dim + j];
        }
    */
    timer_stop(TMR_GRAPHSUM_FW);
}

void GraphSum::backward() {
    timer_start(TMR_GRAPHSUM_BW);
    cuda_in->zero_grad();
    // GPU blocks and threads settings
    dim3 blocksPerGrid((cuda_graph->indptr_size - 1 + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, (dim + MAX_THREADS_PER_BLOCK_2D - 1) / MAX_THREADS_PER_BLOCK_2D, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_2D, MAX_THREADS_PER_BLOCK_2D, 1);
    // Launch kernel
    // SAME EXACT CODE STRUCTURE AS GRAPHSUM FORWARD, BUT WITH GRADIENTS AND WITH IN AND OUT SWAPPED!
    graphsum_forward_parallel<<<blocksPerGrid, threadsPerBlock>>>(cuda_out->grad, cuda_in->grad, cuda_graph->indptr, cuda_graph->indices, cuda_graph->indptr_size - 1, dim);
    check_kernel_call();
    hipDeviceSynchronize();
    /*
    in->zero_grad();
    for (int src = 0; src < graph->indptr.size() - 1; src++)
        for (int i = graph->indptr[src]; i < graph->indptr[src + 1]; i++) {
            int dst = graph->indices[i];
            float coef = 1.0 / sqrtf(
                    (graph->indptr[src + 1] - graph->indptr[src]) * (graph->indptr[dst + 1] - graph->indptr[dst])
            );
            for (int j = 0; j < dim; j++)
                in->grad[src * dim + j] += coef * out->grad[dst * dim + j];
        }
    */
    timer_stop(TMR_GRAPHSUM_BW);
}

// ################################################################################################################

/**
 * Each predicted class probability is compared to the actual class desired and a loss is computed to penalize the proabability based on how far it is with respect to the actual expected value.
 * Also called logaritmic loss. 
*/
CrossEntropyLoss::CrossEntropyLoss(CudaVariable *cuda_logits, int *cuda_truth, float *loss, float *cuda_loss, int num_classes) :
        cuda_logits(cuda_logits), cuda_truth(cuda_truth), loss(loss), cuda_loss(cuda_loss), num_classes(num_classes) {
            // loss in CrossEntropyLoss loss is a pointer pointing to the loss value in GCN
            // cuda_loss in CrossEntropyLoss is a pointer pointing to the same GPU memory area pointed by cuda_loss in GCN
        }

__global__ void crossentropyloss_forward_parallel1(bool training, int *truth, float *logits_data, float *logits_grad, float *total_loss, int *count, int N, int n) {
    // N: logits->data.size(), n: num_classes
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N / n) {
        if (truth[i] >= 0) {
            atomicAdd(&(*count), 1);
            float *logit = &logits_data[i * n]; // each thread works on a different chunk of logits_data
            float max_logit = -1e30, sum_exp = 0.0f;
            for (int j = 0; j < n; j++)
                max_logit = fmaxf(max_logit, logit[j]);
            for (int j = 0; j < n; j++) {
                logit[j] -= max_logit;
                sum_exp += expf(logit[j]);
            }
            atomicAdd(&(*total_loss), logf(sum_exp) - logit[truth[i]]);
            if (training) {
                for (int j = 0; j < n; j++) {
                    float prob = expf(logit[j]) / sum_exp;
                    logits_grad[i * n + j] = prob; // each thread works on a different chunk of logits_grad
                }
                __syncthreads();
                atomicAdd(&logits_grad[i * n + truth[i]], -1.0f);
            }
        }
    }
}

__global__ void crossentropyloss_forward_parallel2(float *loss, float *total_loss, int *count) {
    *loss = *total_loss / *count;
}

__global__ void crossentropyloss_forward_parallel3(float *logits_grad, int *count, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N)
        logits_grad[i] /= *count;
}

void CrossEntropyLoss::forward(bool training) {
    timer_start(TMR_LOSS_FW);
    float total_loss = 0.0f;
    float *cuda_total_loss;
    check_call(hipMalloc(&cuda_total_loss, sizeof(float)));
    check_call(hipMemcpy(cuda_total_loss, &total_loss, sizeof(float), hipMemcpyHostToDevice));
    int count = 0;
    int *cuda_count;
    check_call(hipMalloc(&cuda_count, sizeof(int)));
    check_call(hipMemcpy(cuda_count, &count, sizeof(int), hipMemcpyHostToDevice));
    if (training) cuda_logits->zero_grad();
    // GPU blocks and threads settings
    dim3 blocksPerGrid1((cuda_logits->size / num_classes + MAX_THREADS_PER_BLOCK_1D - 1) / MAX_THREADS_PER_BLOCK_1D, 1, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_1D, 1, 1);
    crossentropyloss_forward_parallel1<<<blocksPerGrid1, threadsPerBlock>>>(training, cuda_truth, cuda_logits->data, cuda_logits->grad, cuda_total_loss, cuda_count, cuda_logits->size, num_classes);
    check_kernel_call();
    hipDeviceSynchronize();
    crossentropyloss_forward_parallel2<<<1, 1>>>(cuda_loss, cuda_total_loss, cuda_count);
    check_kernel_call();
    hipDeviceSynchronize();
    check_call(hipMemcpy(&(*loss), cuda_loss, sizeof(float), hipMemcpyDeviceToHost));
    if (training) {
        dim3 blocksPerGrid3((cuda_logits->size + MAX_THREADS_PER_BLOCK_1D - 1) / MAX_THREADS_PER_BLOCK_1D, 1, 1);
        crossentropyloss_forward_parallel3<<<blocksPerGrid3, threadsPerBlock>>>(cuda_logits->grad, cuda_count, cuda_logits->size);
        check_kernel_call();
        hipDeviceSynchronize();
    }
    check_call(hipFree(cuda_total_loss));
    check_call(hipFree(cuda_count));
    /*
    float total_loss = 0;
    int count = 0;
    if (training) logits->zero_grad();
    for (int i = 0; i < logits->data.size() / num_classes; i++) {
        if (truth[i] < 0) continue;
        count++;
        float *logit = &logits->data[i * num_classes];
        float max_logit = -1e30, sum_exp = 0;
        for (int j = 0; j < num_classes; j++)
            max_logit = fmax(max_logit, logit[j]);
        for (int j = 0; j < num_classes; j++) {
            logit[j] -= max_logit;
            sum_exp += expf(logit[j]);
        }
        total_loss += logf(sum_exp) - logit[truth[i]];

        if (training) {
            for (int j = 0; j < num_classes; j++) {
                float prob = expf(logit[j]) / sum_exp;
                logits->grad[i * num_classes + j] = prob;
            }
            logits->grad[i * num_classes + truth[i]] -= 1.0;
        }
    }
    *loss = total_loss / count;
    if (training)
        for (float & i : logits->grad)
            i /= count;
    */
    timer_stop(TMR_LOSS_FW);
}

void CrossEntropyLoss::backward() {}

// ################################################################################################################

/**
 * Rectified Linear Unit activation function.
 * If input is negative it will output 0.
*/
ReLU::ReLU(CudaVariable *cuda_in) {
    this->cuda_in = cuda_in;
    check_call(hipMalloc(&cuda_mask, cuda_in->size * sizeof(bool)));
}

ReLU::~ReLU() {
    std::cout << "Deallocating ReLU." << std::endl;
    check_call(hipFree(cuda_mask));
}

__global__ void relu_forward_parallel(float *in, bool *mask, int N, bool training) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        bool keep = in[i] > 0;
        if (training) mask[i] = keep;
        if (!keep) in[i] = 0.0f;
    }
}

void ReLU::forward(bool training) {
    timer_start(TMR_RELU_FW);
    // GPU blocks and threads settings
    dim3 blocksPerGrid((cuda_in->size + MAX_THREADS_PER_BLOCK_1D - 1) / MAX_THREADS_PER_BLOCK_1D, 1, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_1D, 1, 1);
    // Launch kernel
    relu_forward_parallel<<<blocksPerGrid, threadsPerBlock>>>(cuda_in->data, cuda_mask, cuda_in->size, training);
    check_kernel_call();
    hipDeviceSynchronize();
    /*
    for (int i = 0; i < in->data.size(); i++) {
        bool keep = in->data[i] > 0;
        if (training) mask[i] = keep;
        if (!keep) in->data[i] = 0;
    }
    */
    timer_stop(TMR_RELU_FW);
}

__global__ void relu_backward_parallel(float *grad, bool *mask, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        if (!mask[i]) grad[i] = 0.0f;
    }
}

void ReLU::backward() {
    timer_start(TMR_RELU_BW);
    // GPU blocks and threads settings
    dim3 blocksPerGrid((cuda_in->size + MAX_THREADS_PER_BLOCK_1D - 1) / MAX_THREADS_PER_BLOCK_1D, 1, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_1D, 1, 1);
    relu_backward_parallel<<<blocksPerGrid, threadsPerBlock>>>(cuda_in->grad, cuda_mask, cuda_in->size);
    check_kernel_call();
    hipDeviceSynchronize();
    /*
    for (int i = 0; i < in->data.size(); i++)
        if (!mask[i]) in->grad[i] = 0;
    */
    timer_stop(TMR_RELU_BW);
}

// ################################################################################################################

/**
 * The dropout layer randomly sets input units to 0 with a frequency of P at each step during training time to prevent overfitting. 
 * Inputs that are not set to 0 are scaled up by 1/(1-P).
*/
Dropout::Dropout(CudaVariable *cuda_in, float p) {
    this->cuda_in = cuda_in;
    this->p = p;
    if (cuda_in->grad)
        check_call(hipMalloc(&cuda_mask, cuda_in->size * sizeof(int)));
    else
        cuda_mask = nullptr;
    // GPU blocks and threads settings
    dim3 blocksPerGrid((cuda_in->size + MAX_THREADS_PER_BLOCK_1D - 1) / MAX_THREADS_PER_BLOCK_1D, 1, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_1D, 1, 1);
    // Initialize CUDA random
    check_call(hipMalloc(&cuda_rand_state, cuda_in->size * sizeof(hiprandState)));
    rand_setup_kernel<<<blocksPerGrid, threadsPerBlock>>>(cuda_rand_state, cuda_in->size);
    check_kernel_call();
    hipDeviceSynchronize();
}

Dropout::~Dropout() {
    std::cout << "Deallocating Dropout." << std::endl;
    if (cuda_mask) check_call(hipFree(cuda_mask));
}

__global__ void dropout_forward_parallel(float *in, int* mask, int N, const int threshold, float scale, hiprandState *rand_state, unsigned rand_max) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        float my_randf = rand_max * hiprand_uniform(&rand_state[i]);
        int my_rand = (int) truncf(my_randf);
        bool keep = my_rand >= threshold;
        in[i] *= keep ? scale : 0;
        if (mask) mask[i] = keep;
    }
}

void Dropout::forward(bool training) {
    if (!training) return;
    timer_start(TMR_DROPOUT_FW);
    const int threshold = int(p * MY_RAND_MAX);
    float scale = 1 / (1 - p);
    // GPU blocks and threads settings
    dim3 blocksPerGrid((cuda_in->size + MAX_THREADS_PER_BLOCK_1D - 1) / MAX_THREADS_PER_BLOCK_1D, 1, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_1D, 1, 1);
    // Launch kernel
    dropout_forward_parallel<<<blocksPerGrid, threadsPerBlock>>>(cuda_in->data, cuda_mask, cuda_in->size, threshold, scale, cuda_rand_state, MY_RAND_MAX);
    check_kernel_call();
    hipDeviceSynchronize();
    /*
    for (int i = 0; i < in->data.size(); i++) {
        bool keep = (int) RAND() >= threshold;
        in->data[i] *= keep ? scale : 0;
        if (mask) mask[i] = keep;
    }
    */
    timer_stop(TMR_DROPOUT_FW);
}

__global__ void dropout_backward_parallel(float *grad, int *mask, int N, float scale) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        grad[i] *= mask[i] ? scale : 0.0f;
    }
}

void Dropout::backward() {
    if (!cuda_mask) return;
    timer_start(TMR_DROPOUT_BW);
    float scale = 1 / (1 - p);
    // GPU blocks and threads settings
    dim3 blocksPerGrid((cuda_in->size + MAX_THREADS_PER_BLOCK_1D - 1) / MAX_THREADS_PER_BLOCK_1D, 1, 1);
    dim3 threadsPerBlock(MAX_THREADS_PER_BLOCK_1D, 1, 1);
    // Launch kernel
    dropout_backward_parallel<<<blocksPerGrid, threadsPerBlock>>>(cuda_in->grad, cuda_mask, cuda_in->size, scale);
    check_kernel_call();
    hipDeviceSynchronize();
    /*
    for (int i = 0; i < in->data.size(); i++)
        in->grad[i] *= mask[i] ? scale : 0;
    */
    timer_stop(TMR_DROPOUT_BW);
}

// ################################################################################################################