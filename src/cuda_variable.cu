#include "hip/hip_runtime.h"
#include "../include/cuda_variable.h"
#include "../include/cuda_check.h"
#include "../include/cuda_rand.h"
#include <cmath>
#include <cstdio>
#include <algorithm>

#define MAX_NUM_THREADS 1024

CudaVariable::CudaVariable(int size, bool requires_grad, bool thread_local_grad): size(size) {
    check_call(hipMalloc(&data, size * sizeof(float)));
    if (requires_grad)
        check_call(hipMalloc(&grad, size * sizeof(float)));
}

CudaVariable::~CudaVariable() {
    //check_call(hipFree(data));
    //check_call(hipFree(grad));
    //check_call(hipFree(local_grad));
}

/**
 * Glorot (Xavier) method for weights initialization
 * WHAT IS WRONG WITH THE PARALLEL VERSION? CHANGE SEED FOR EVERY ALLOCATED WEIGHT!
*/

__global__ void glorot_parallel(float *data, float range, int size, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
        data[i] = (hiprand_uniform(&rand_state[i]) / MY_RAND_MAX - 0.5) * range * 2;
}

void CudaVariable::glorot(int in_size, int out_size) {
    hiprandState *cuda_rand_state;
    // Initialize CUDA random
    check_call(hipMalloc(&cuda_rand_state, size * sizeof(hiprandState)));
    rand_setup_kernel<<<(size + MAX_NUM_THREADS - 1) / MAX_NUM_THREADS, MAX_NUM_THREADS>>>(cuda_rand_state, size, in_size + out_size);
    check_kernel_call();
    hipDeviceSynchronize();
    float range = sqrtf(6.0f / (in_size + out_size)); 
    glorot_parallel<<<(size + MAX_NUM_THREADS - 1) / MAX_NUM_THREADS, MAX_NUM_THREADS>>>(data, range, size, cuda_rand_state);
    check_kernel_call();
    hipDeviceSynchronize();
    /*
    float range = sqrtf(6.0f / (in_size + out_size));
    float *temp_data = (float *) malloc(size * sizeof(float));
    for(int i = 0; i < size; ++i)
        temp_data[i] = (float(RAND()) / MY_RAND_MAX - 0.5) * range * 2;
    check_call(hipMemcpy(data, temp_data, size * sizeof(float), hipMemcpyHostToDevice));
    free(temp_data);
    */
}

__global__ void zero_parallel(float *data, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
        data[i] = 0.0f;
}

void CudaVariable::zero() {
    zero_parallel<<<(size + MAX_NUM_THREADS - 1) / MAX_NUM_THREADS, MAX_NUM_THREADS>>>(data, size);
    check_kernel_call();
    hipDeviceSynchronize();
}

void CudaVariable::zero_grad() {
    zero_parallel<<<(size + MAX_NUM_THREADS - 1) / MAX_NUM_THREADS, MAX_NUM_THREADS>>>(grad, size);
    check_kernel_call();
    hipDeviceSynchronize();
}

// Reduction via warps
__device__ float warp_reduce(float val) {
    int warp_size = 32;
    for (int offset = warp_size / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

__global__ void grad_norm_parallel(float *in, float *out, int size) {
    int warp_size = 32;
    float sum = 0.0f;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
        sum += in[i] * in[i];
    sum = warp_reduce(sum);
    if ((threadIdx.x & (warp_size - 1)) == 0)
        atomicAdd(out, sum);
}

float CudaVariable::grad_norm() {
    float norm;
    float *cuda_norm;
    check_call(hipMalloc(&cuda_norm, sizeof(float)));
    grad_norm_parallel<<<(size + MAX_NUM_THREADS - 1) / MAX_NUM_THREADS, MAX_NUM_THREADS>>>(grad, cuda_norm, size);
    check_kernel_call();
    hipDeviceSynchronize();
    check_call(hipMemcpy(&norm, cuda_norm, sizeof(float), hipMemcpyDeviceToHost));
    check_call(hipFree(cuda_norm));
    return sqrtf(norm);
}