#include "hip/hip_runtime.h"
#include "../include/cuda_rand.h"

__global__ void setup_kernel(hiprandState *state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // hiprand_init (unsigned long long seed, unsigned long long sequence, unsigned long long offset, hiprandState_t *state)
    hiprand_init(1234, i, 0, &state[i]);
}