#include "hip/hip_runtime.h"
#include "../include/cuda_rand.h"

__global__ void rand_setup_kernel(hiprandState *state, int seed, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N)
        // hiprand_init (unsigned long long seed, unsigned long long sequence, unsigned long long offset, hiprandState_t *state)
        hiprand_init(seed, i, 0, &state[i]);
}